#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "thrust.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            thrust::host_vector<int> h_in(idata, idata + n);


            thrust::device_vector<int> d_in = h_in;
            thrust::device_vector<int> d_out(n);
            timer().startGpuTimer();

            thrust::exclusive_scan(d_in.begin(), d_in.end(), d_out.begin());

            timer().endGpuTimer();
            thrust::copy(d_out.begin(), d_out.end(), odata);
            
        }
    }
}
